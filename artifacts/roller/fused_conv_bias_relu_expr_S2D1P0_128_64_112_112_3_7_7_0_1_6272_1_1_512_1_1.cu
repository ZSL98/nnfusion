#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

//full_dimensions: [64, 1605632, 147]

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv_unpad, float* __restrict__ bias) {
  float conv_local[32];
  __shared__ float data_pad_shared[8192];
  __shared__ float kernel_pad_shared[2048];
  float data_pad_shared_local[4];
  float kernel_pad_shared_local[8];
  conv_local[0] = 0.000000e+00f;
  conv_local[4] = 0.000000e+00f;
  conv_local[8] = 0.000000e+00f;
  conv_local[12] = 0.000000e+00f;
  conv_local[16] = 0.000000e+00f;
  conv_local[20] = 0.000000e+00f;
  conv_local[24] = 0.000000e+00f;
  conv_local[28] = 0.000000e+00f;
  conv_local[1] = 0.000000e+00f;
  conv_local[5] = 0.000000e+00f;
  conv_local[9] = 0.000000e+00f;
  conv_local[13] = 0.000000e+00f;
  conv_local[17] = 0.000000e+00f;
  conv_local[21] = 0.000000e+00f;
  conv_local[25] = 0.000000e+00f;
  conv_local[29] = 0.000000e+00f;
  conv_local[2] = 0.000000e+00f;
  conv_local[6] = 0.000000e+00f;
  conv_local[10] = 0.000000e+00f;
  conv_local[14] = 0.000000e+00f;
  conv_local[18] = 0.000000e+00f;
  conv_local[22] = 0.000000e+00f;
  conv_local[26] = 0.000000e+00f;
  conv_local[30] = 0.000000e+00f;
  conv_local[3] = 0.000000e+00f;
  conv_local[7] = 0.000000e+00f;
  conv_local[11] = 0.000000e+00f;
  conv_local[15] = 0.000000e+00f;
  conv_local[19] = 0.000000e+00f;
  conv_local[23] = 0.000000e+00f;
  conv_local[27] = 0.000000e+00f;
  conv_local[31] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 5; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[((int)threadIdx.x)] = data[(((((((((int)blockIdx.x) / 49) * 158700) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + (((ra_fused0_outer * 4) + (((int)threadIdx.x) >> 8)) % 7))];
    data_pad_shared[(((int)threadIdx.x) + 512)] = data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 2) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 2) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 2) % 7))];
    data_pad_shared[(((int)threadIdx.x) + 1024)] = data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 4) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 4) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 4) % 7))];
    data_pad_shared[(((int)threadIdx.x) + 1536)] = data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 6) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 6) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 6) % 7))];
    data_pad_shared[(((int)threadIdx.x) + 2048)] = data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 8) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 8) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 1) % 7))];
    data_pad_shared[(((int)threadIdx.x) + 2560)] = data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 10) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 10) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 3) % 7))];
    data_pad_shared[(((int)threadIdx.x) + 3072)] = data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 12) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 12) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 5) % 7))];
    data_pad_shared[(((int)threadIdx.x) + 3584)] = data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 14) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) / 7) + 2) % 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + (((ra_fused0_outer * 4) + (((int)threadIdx.x) >> 8)) % 7))];
    data_pad_shared[(((int)threadIdx.x) + 4096)] = data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 16) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 16) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 2) % 7))];
    data_pad_shared[(((int)threadIdx.x) + 4608)] = ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) < 129) ? data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 18) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 18) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 4) % 7))] : 0.000000e+00f);
    data_pad_shared[(((int)threadIdx.x) + 5120)] = ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) < 127) ? data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 20) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 20) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 6) % 7))] : 0.000000e+00f);
    data_pad_shared[(((int)threadIdx.x) + 5632)] = ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) < 125) ? data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 22) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 22) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 1) % 7))] : 0.000000e+00f);
    data_pad_shared[(((int)threadIdx.x) + 6144)] = ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) < 123) ? data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 24) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 24) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 3) % 7))] : 0.000000e+00f);
    data_pad_shared[(((int)threadIdx.x) + 6656)] = ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) < 121) ? data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 26) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 26) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 5) % 7))] : 0.000000e+00f);
    data_pad_shared[(((int)threadIdx.x) + 7168)] = ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) < 119) ? data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 28) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) / 7) + 4) % 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + (((ra_fused0_outer * 4) + (((int)threadIdx.x) >> 8)) % 7))] : 0.000000e+00f);
    data_pad_shared[(((int)threadIdx.x) + 7680)] = ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) < 117) ? data[(((((((((int)blockIdx.x) / 49) * 158700) + (((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 30) / 49) * 52900)) + (((((((int)blockIdx.x) % 49) * 16) + ((((int)threadIdx.x) & 255) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 30) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) & 255)) % 112) * 2)) + ((((ra_fused0_outer * 32) + (((int)threadIdx.x) >> 8)) + 2) % 7))] : 0.000000e+00f);
    kernel_pad_shared[((int)threadIdx.x)] = ((((ra_fused0_outer * 32) + (((int)threadIdx.x) & 31)) < 147) ? kernel[((((((int)threadIdx.x) >> 5) * 147) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31))] : 0.000000e+00f);
    kernel_pad_shared[(((int)threadIdx.x) + 512)] = ((((ra_fused0_outer * 32) + (((int)threadIdx.x) & 31)) < 147) ? kernel[(((((((int)threadIdx.x) >> 5) * 147) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 2352)] : 0.000000e+00f);
    kernel_pad_shared[(((int)threadIdx.x) + 1024)] = ((((ra_fused0_outer * 32) + (((int)threadIdx.x) & 31)) < 147) ? kernel[(((((((int)threadIdx.x) >> 5) * 147) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 4704)] : 0.000000e+00f);
    kernel_pad_shared[(((int)threadIdx.x) + 1536)] = ((((ra_fused0_outer * 32) + (((int)threadIdx.x) & 31)) < 147) ? kernel[(((((((int)threadIdx.x) >> 5) * 147) + (ra_fused0_outer * 32)) + (((int)threadIdx.x) & 31)) + 7056)] : 0.000000e+00f);
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 32; ++ra_fused0_inner_outer) {
      data_pad_shared_local[0] = data_pad_shared[((ra_fused0_inner_outer * 256) + (((int)threadIdx.x) & 63))];
      data_pad_shared_local[1] = data_pad_shared[(((ra_fused0_inner_outer * 256) + (((int)threadIdx.x) & 63)) + 64)];
      data_pad_shared_local[2] = data_pad_shared[(((ra_fused0_inner_outer * 256) + (((int)threadIdx.x) & 63)) + 128)];
      data_pad_shared_local[3] = data_pad_shared[(((ra_fused0_inner_outer * 256) + (((int)threadIdx.x) & 63)) + 192)];
      kernel_pad_shared_local[0] = kernel_pad_shared[(((((int)threadIdx.x) >> 6) * 32) + ra_fused0_inner_outer)];
      kernel_pad_shared_local[1] = kernel_pad_shared[((((((int)threadIdx.x) >> 6) * 32) + ra_fused0_inner_outer) + 256)];
      kernel_pad_shared_local[2] = kernel_pad_shared[((((((int)threadIdx.x) >> 6) * 32) + ra_fused0_inner_outer) + 512)];
      kernel_pad_shared_local[3] = kernel_pad_shared[((((((int)threadIdx.x) >> 6) * 32) + ra_fused0_inner_outer) + 768)];
      kernel_pad_shared_local[4] = kernel_pad_shared[((((((int)threadIdx.x) >> 6) * 32) + ra_fused0_inner_outer) + 1024)];
      kernel_pad_shared_local[5] = kernel_pad_shared[((((((int)threadIdx.x) >> 6) * 32) + ra_fused0_inner_outer) + 1280)];
      kernel_pad_shared_local[6] = kernel_pad_shared[((((((int)threadIdx.x) >> 6) * 32) + ra_fused0_inner_outer) + 1536)];
      kernel_pad_shared_local[7] = kernel_pad_shared[((((((int)threadIdx.x) >> 6) * 32) + ra_fused0_inner_outer) + 1792)];
      if (((ra_fused0_outer * 32) + ra_fused0_inner_outer) < 147) {
        conv_local[0] = (conv_local[0] + (data_pad_shared_local[0] * kernel_pad_shared_local[0]));
        conv_local[4] = (conv_local[4] + (data_pad_shared_local[0] * kernel_pad_shared_local[1]));
        conv_local[8] = (conv_local[8] + (data_pad_shared_local[0] * kernel_pad_shared_local[2]));
        conv_local[12] = (conv_local[12] + (data_pad_shared_local[0] * kernel_pad_shared_local[3]));
        conv_local[16] = (conv_local[16] + (data_pad_shared_local[0] * kernel_pad_shared_local[4]));
        conv_local[20] = (conv_local[20] + (data_pad_shared_local[0] * kernel_pad_shared_local[5]));
        conv_local[24] = (conv_local[24] + (data_pad_shared_local[0] * kernel_pad_shared_local[6]));
        conv_local[28] = (conv_local[28] + (data_pad_shared_local[0] * kernel_pad_shared_local[7]));
        conv_local[1] = (conv_local[1] + (data_pad_shared_local[1] * kernel_pad_shared_local[0]));
        conv_local[5] = (conv_local[5] + (data_pad_shared_local[1] * kernel_pad_shared_local[1]));
        conv_local[9] = (conv_local[9] + (data_pad_shared_local[1] * kernel_pad_shared_local[2]));
        conv_local[13] = (conv_local[13] + (data_pad_shared_local[1] * kernel_pad_shared_local[3]));
        conv_local[17] = (conv_local[17] + (data_pad_shared_local[1] * kernel_pad_shared_local[4]));
        conv_local[21] = (conv_local[21] + (data_pad_shared_local[1] * kernel_pad_shared_local[5]));
        conv_local[25] = (conv_local[25] + (data_pad_shared_local[1] * kernel_pad_shared_local[6]));
        conv_local[29] = (conv_local[29] + (data_pad_shared_local[1] * kernel_pad_shared_local[7]));
        conv_local[2] = (conv_local[2] + (data_pad_shared_local[2] * kernel_pad_shared_local[0]));
        conv_local[6] = (conv_local[6] + (data_pad_shared_local[2] * kernel_pad_shared_local[1]));
        conv_local[10] = (conv_local[10] + (data_pad_shared_local[2] * kernel_pad_shared_local[2]));
        conv_local[14] = (conv_local[14] + (data_pad_shared_local[2] * kernel_pad_shared_local[3]));
        conv_local[18] = (conv_local[18] + (data_pad_shared_local[2] * kernel_pad_shared_local[4]));
        conv_local[22] = (conv_local[22] + (data_pad_shared_local[2] * kernel_pad_shared_local[5]));
        conv_local[26] = (conv_local[26] + (data_pad_shared_local[2] * kernel_pad_shared_local[6]));
        conv_local[30] = (conv_local[30] + (data_pad_shared_local[2] * kernel_pad_shared_local[7]));
        conv_local[3] = (conv_local[3] + (data_pad_shared_local[3] * kernel_pad_shared_local[0]));
        conv_local[7] = (conv_local[7] + (data_pad_shared_local[3] * kernel_pad_shared_local[1]));
        conv_local[11] = (conv_local[11] + (data_pad_shared_local[3] * kernel_pad_shared_local[2]));
        conv_local[15] = (conv_local[15] + (data_pad_shared_local[3] * kernel_pad_shared_local[3]));
        conv_local[19] = (conv_local[19] + (data_pad_shared_local[3] * kernel_pad_shared_local[4]));
        conv_local[23] = (conv_local[23] + (data_pad_shared_local[3] * kernel_pad_shared_local[5]));
        conv_local[27] = (conv_local[27] + (data_pad_shared_local[3] * kernel_pad_shared_local[6]));
        conv_local[31] = (conv_local[31] + (data_pad_shared_local[3] * kernel_pad_shared_local[7]));
      }
    }
  }
  conv_unpad[((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63))] = max((conv_local[0] + bias[(((int)threadIdx.x) >> 6)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 12845056)] = max((conv_local[4] + bias[((((int)threadIdx.x) >> 6) + 8)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 25690112)] = max((conv_local[8] + bias[((((int)threadIdx.x) >> 6) + 16)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 38535168)] = max((conv_local[12] + bias[((((int)threadIdx.x) >> 6) + 24)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 51380224)] = max((conv_local[16] + bias[((((int)threadIdx.x) >> 6) + 32)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 64225280)] = max((conv_local[20] + bias[((((int)threadIdx.x) >> 6) + 40)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 77070336)] = max((conv_local[24] + bias[((((int)threadIdx.x) >> 6) + 48)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 89915392)] = max((conv_local[28] + bias[((((int)threadIdx.x) >> 6) + 56)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 64)] = max((conv_local[1] + bias[(((int)threadIdx.x) >> 6)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 12845120)] = max((conv_local[5] + bias[((((int)threadIdx.x) >> 6) + 8)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 25690176)] = max((conv_local[9] + bias[((((int)threadIdx.x) >> 6) + 16)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 38535232)] = max((conv_local[13] + bias[((((int)threadIdx.x) >> 6) + 24)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 51380288)] = max((conv_local[17] + bias[((((int)threadIdx.x) >> 6) + 32)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 64225344)] = max((conv_local[21] + bias[((((int)threadIdx.x) >> 6) + 40)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 77070400)] = max((conv_local[25] + bias[((((int)threadIdx.x) >> 6) + 48)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 89915456)] = max((conv_local[29] + bias[((((int)threadIdx.x) >> 6) + 56)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 128)] = max((conv_local[2] + bias[(((int)threadIdx.x) >> 6)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 12845184)] = max((conv_local[6] + bias[((((int)threadIdx.x) >> 6) + 8)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 25690240)] = max((conv_local[10] + bias[((((int)threadIdx.x) >> 6) + 16)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 38535296)] = max((conv_local[14] + bias[((((int)threadIdx.x) >> 6) + 24)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 51380352)] = max((conv_local[18] + bias[((((int)threadIdx.x) >> 6) + 32)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 64225408)] = max((conv_local[22] + bias[((((int)threadIdx.x) >> 6) + 40)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 77070464)] = max((conv_local[26] + bias[((((int)threadIdx.x) >> 6) + 48)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 89915520)] = max((conv_local[30] + bias[((((int)threadIdx.x) >> 6) + 56)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 192)] = max((conv_local[3] + bias[(((int)threadIdx.x) >> 6)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 12845248)] = max((conv_local[7] + bias[((((int)threadIdx.x) >> 6) + 8)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 25690304)] = max((conv_local[11] + bias[((((int)threadIdx.x) >> 6) + 16)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 38535360)] = max((conv_local[15] + bias[((((int)threadIdx.x) >> 6) + 24)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 51380416)] = max((conv_local[19] + bias[((((int)threadIdx.x) >> 6) + 32)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 64225472)] = max((conv_local[23] + bias[((((int)threadIdx.x) >> 6) + 40)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 77070528)] = max((conv_local[27] + bias[((((int)threadIdx.x) >> 6) + 48)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 6) * 1605632) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) & 63)) + 89915584)] = max((conv_local[31] + bias[((((int)threadIdx.x) >> 6) + 56)]), 0.000000e+00f);
}


int main(int argc, char *argv[])
{
    std::string path;
    int input_size0 = 20313600;
    int input_size1 = 9408;
    int input_size2 = 64;
    int output_size0 = 102760448;

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *input0h, *input1h, *input2h, *output0h;
    float *input0d, *input1d, *input2d, *output0d;
    input0h = (float*)malloc(81254400);
    input1h = (float*)malloc(37632);
    input2h = (float*)malloc(256);

    hipMalloc((void **)&input0d, 81254400);
    hipMalloc((void **)&input1d, 37632);
    hipMalloc((void **)&input2d, 256);
    hipMalloc((void **)&output0d, 411041792);

    srand(1);
    for (int i = 0; i < input_size0; ++ i)
        input0h[i] = 1;
    for (int i = 0; i < input_size1; ++ i)
        input1h[i] = 1;
    for (int i = 0; i < input_size2; ++ i)
        input2h[i] = 1;

    hipMemcpy(input0d, input0h, 81254400, hipMemcpyHostToDevice);
    hipMemcpy(input1d, input1h, 37632, hipMemcpyHostToDevice);
    hipMemcpy(input2d, input2h, 256, hipMemcpyHostToDevice);

    dim3 grid(6272, 1, 1);
    dim3 block(512, 1, 1);

    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)input0d, (float*)input1d, (float*)output0d, (float*)input2d);
        hipDeviceSynchronize();
    }
}
