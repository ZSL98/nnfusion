#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cu_helper.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <string>

//full_dimensions: [64, 1605632, 147]

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(256) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ conv_unpad, float* __restrict__ bias) {
  float conv_local[32];
  __shared__ float data_pad_shared[1024];
  __shared__ float kernel_pad_shared[512];
  float data_pad_shared_local[4];
  float kernel_pad_shared_local[8];
  conv_local[0] = 0.000000e+00f;
  conv_local[4] = 0.000000e+00f;
  conv_local[8] = 0.000000e+00f;
  conv_local[12] = 0.000000e+00f;
  conv_local[16] = 0.000000e+00f;
  conv_local[20] = 0.000000e+00f;
  conv_local[24] = 0.000000e+00f;
  conv_local[28] = 0.000000e+00f;
  conv_local[1] = 0.000000e+00f;
  conv_local[5] = 0.000000e+00f;
  conv_local[9] = 0.000000e+00f;
  conv_local[13] = 0.000000e+00f;
  conv_local[17] = 0.000000e+00f;
  conv_local[21] = 0.000000e+00f;
  conv_local[25] = 0.000000e+00f;
  conv_local[29] = 0.000000e+00f;
  conv_local[2] = 0.000000e+00f;
  conv_local[6] = 0.000000e+00f;
  conv_local[10] = 0.000000e+00f;
  conv_local[14] = 0.000000e+00f;
  conv_local[18] = 0.000000e+00f;
  conv_local[22] = 0.000000e+00f;
  conv_local[26] = 0.000000e+00f;
  conv_local[30] = 0.000000e+00f;
  conv_local[3] = 0.000000e+00f;
  conv_local[7] = 0.000000e+00f;
  conv_local[11] = 0.000000e+00f;
  conv_local[15] = 0.000000e+00f;
  conv_local[19] = 0.000000e+00f;
  conv_local[23] = 0.000000e+00f;
  conv_local[27] = 0.000000e+00f;
  conv_local[31] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 19; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[((int)threadIdx.x)] = data[(((((((((int)blockIdx.x) / 98) * 158700) + ((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) / 49) * 52900)) + (((((((int)blockIdx.x) % 98) * 8) + ((((int)threadIdx.x) & 127) >> 4)) / 7) * 460)) + (((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) & 127)) % 112) * 2)) + (((((int)threadIdx.x) >> 7) + ra_fused0_outer) % 7))];
    data_pad_shared[(((int)threadIdx.x) + 256)] = ((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) < 145) ? data[(((((((((int)blockIdx.x) / 98) * 158700) + (((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) + 2) / 49) * 52900)) + (((((((int)blockIdx.x) % 98) * 8) + ((((int)threadIdx.x) & 127) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) + 2) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) & 127)) % 112) * 2)) + ((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) + 2) % 7))] : 0.000000e+00f);
    data_pad_shared[(((int)threadIdx.x) + 512)] = ((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) < 143) ? data[(((((((((int)blockIdx.x) / 98) * 158700) + (((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) + 4) / 49) * 52900)) + (((((((int)blockIdx.x) % 98) * 8) + ((((int)threadIdx.x) & 127) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) + 4) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) & 127)) % 112) * 2)) + ((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) + 4) % 7))] : 0.000000e+00f);
    data_pad_shared[(((int)threadIdx.x) + 768)] = ((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) < 141) ? data[(((((((((int)blockIdx.x) / 98) * 158700) + (((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) + 6) / 49) * 52900)) + (((((((int)blockIdx.x) % 98) * 8) + ((((int)threadIdx.x) & 127) >> 4)) / 7) * 460)) + ((((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) + 6) % 49) / 7) * 230)) + ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) & 127)) % 112) * 2)) + ((((ra_fused0_outer * 8) + (((int)threadIdx.x) >> 7)) + 6) % 7))] : 0.000000e+00f);
    kernel_pad_shared[((int)threadIdx.x)] = ((((ra_fused0_outer * 8) + (((int)threadIdx.x) & 7)) < 147) ? kernel[((((((int)threadIdx.x) >> 3) * 147) + (ra_fused0_outer * 8)) + (((int)threadIdx.x) & 7))] : 0.000000e+00f);
    kernel_pad_shared[(((int)threadIdx.x) + 256)] = ((((ra_fused0_outer * 8) + (((int)threadIdx.x) & 7)) < 147) ? kernel[(((((((int)threadIdx.x) >> 3) * 147) + (ra_fused0_outer * 8)) + (((int)threadIdx.x) & 7)) + 4704)] : 0.000000e+00f);
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 8; ++ra_fused0_inner_outer) {
      data_pad_shared_local[0] = data_pad_shared[((ra_fused0_inner_outer * 128) + (((int)threadIdx.x) & 31))];
      data_pad_shared_local[1] = data_pad_shared[(((ra_fused0_inner_outer * 128) + (((int)threadIdx.x) & 31)) + 32)];
      data_pad_shared_local[2] = data_pad_shared[(((ra_fused0_inner_outer * 128) + (((int)threadIdx.x) & 31)) + 64)];
      data_pad_shared_local[3] = data_pad_shared[(((ra_fused0_inner_outer * 128) + (((int)threadIdx.x) & 31)) + 96)];
      kernel_pad_shared_local[0] = kernel_pad_shared[(((((int)threadIdx.x) >> 5) * 8) + ra_fused0_inner_outer)];
      kernel_pad_shared_local[1] = kernel_pad_shared[((((((int)threadIdx.x) >> 5) * 8) + ra_fused0_inner_outer) + 64)];
      kernel_pad_shared_local[2] = kernel_pad_shared[((((((int)threadIdx.x) >> 5) * 8) + ra_fused0_inner_outer) + 128)];
      kernel_pad_shared_local[3] = kernel_pad_shared[((((((int)threadIdx.x) >> 5) * 8) + ra_fused0_inner_outer) + 192)];
      kernel_pad_shared_local[4] = kernel_pad_shared[((((((int)threadIdx.x) >> 5) * 8) + ra_fused0_inner_outer) + 256)];
      kernel_pad_shared_local[5] = kernel_pad_shared[((((((int)threadIdx.x) >> 5) * 8) + ra_fused0_inner_outer) + 320)];
      kernel_pad_shared_local[6] = kernel_pad_shared[((((((int)threadIdx.x) >> 5) * 8) + ra_fused0_inner_outer) + 384)];
      kernel_pad_shared_local[7] = kernel_pad_shared[((((((int)threadIdx.x) >> 5) * 8) + ra_fused0_inner_outer) + 448)];
      if (((ra_fused0_outer * 8) + ra_fused0_inner_outer) < 147) {
        conv_local[0] = (conv_local[0] + (data_pad_shared_local[0] * kernel_pad_shared_local[0]));
        conv_local[4] = (conv_local[4] + (data_pad_shared_local[0] * kernel_pad_shared_local[1]));
        conv_local[8] = (conv_local[8] + (data_pad_shared_local[0] * kernel_pad_shared_local[2]));
        conv_local[12] = (conv_local[12] + (data_pad_shared_local[0] * kernel_pad_shared_local[3]));
        conv_local[16] = (conv_local[16] + (data_pad_shared_local[0] * kernel_pad_shared_local[4]));
        conv_local[20] = (conv_local[20] + (data_pad_shared_local[0] * kernel_pad_shared_local[5]));
        conv_local[24] = (conv_local[24] + (data_pad_shared_local[0] * kernel_pad_shared_local[6]));
        conv_local[28] = (conv_local[28] + (data_pad_shared_local[0] * kernel_pad_shared_local[7]));
        conv_local[1] = (conv_local[1] + (data_pad_shared_local[1] * kernel_pad_shared_local[0]));
        conv_local[5] = (conv_local[5] + (data_pad_shared_local[1] * kernel_pad_shared_local[1]));
        conv_local[9] = (conv_local[9] + (data_pad_shared_local[1] * kernel_pad_shared_local[2]));
        conv_local[13] = (conv_local[13] + (data_pad_shared_local[1] * kernel_pad_shared_local[3]));
        conv_local[17] = (conv_local[17] + (data_pad_shared_local[1] * kernel_pad_shared_local[4]));
        conv_local[21] = (conv_local[21] + (data_pad_shared_local[1] * kernel_pad_shared_local[5]));
        conv_local[25] = (conv_local[25] + (data_pad_shared_local[1] * kernel_pad_shared_local[6]));
        conv_local[29] = (conv_local[29] + (data_pad_shared_local[1] * kernel_pad_shared_local[7]));
        conv_local[2] = (conv_local[2] + (data_pad_shared_local[2] * kernel_pad_shared_local[0]));
        conv_local[6] = (conv_local[6] + (data_pad_shared_local[2] * kernel_pad_shared_local[1]));
        conv_local[10] = (conv_local[10] + (data_pad_shared_local[2] * kernel_pad_shared_local[2]));
        conv_local[14] = (conv_local[14] + (data_pad_shared_local[2] * kernel_pad_shared_local[3]));
        conv_local[18] = (conv_local[18] + (data_pad_shared_local[2] * kernel_pad_shared_local[4]));
        conv_local[22] = (conv_local[22] + (data_pad_shared_local[2] * kernel_pad_shared_local[5]));
        conv_local[26] = (conv_local[26] + (data_pad_shared_local[2] * kernel_pad_shared_local[6]));
        conv_local[30] = (conv_local[30] + (data_pad_shared_local[2] * kernel_pad_shared_local[7]));
        conv_local[3] = (conv_local[3] + (data_pad_shared_local[3] * kernel_pad_shared_local[0]));
        conv_local[7] = (conv_local[7] + (data_pad_shared_local[3] * kernel_pad_shared_local[1]));
        conv_local[11] = (conv_local[11] + (data_pad_shared_local[3] * kernel_pad_shared_local[2]));
        conv_local[15] = (conv_local[15] + (data_pad_shared_local[3] * kernel_pad_shared_local[3]));
        conv_local[19] = (conv_local[19] + (data_pad_shared_local[3] * kernel_pad_shared_local[4]));
        conv_local[23] = (conv_local[23] + (data_pad_shared_local[3] * kernel_pad_shared_local[5]));
        conv_local[27] = (conv_local[27] + (data_pad_shared_local[3] * kernel_pad_shared_local[6]));
        conv_local[31] = (conv_local[31] + (data_pad_shared_local[3] * kernel_pad_shared_local[7]));
      }
    }
  }
  conv_unpad[((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31))] = max((conv_local[0] + bias[(((int)threadIdx.x) >> 5)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 12845056)] = max((conv_local[4] + bias[((((int)threadIdx.x) >> 5) + 8)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 25690112)] = max((conv_local[8] + bias[((((int)threadIdx.x) >> 5) + 16)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 38535168)] = max((conv_local[12] + bias[((((int)threadIdx.x) >> 5) + 24)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 51380224)] = max((conv_local[16] + bias[((((int)threadIdx.x) >> 5) + 32)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 64225280)] = max((conv_local[20] + bias[((((int)threadIdx.x) >> 5) + 40)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 77070336)] = max((conv_local[24] + bias[((((int)threadIdx.x) >> 5) + 48)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 89915392)] = max((conv_local[28] + bias[((((int)threadIdx.x) >> 5) + 56)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 32)] = max((conv_local[1] + bias[(((int)threadIdx.x) >> 5)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 12845088)] = max((conv_local[5] + bias[((((int)threadIdx.x) >> 5) + 8)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 25690144)] = max((conv_local[9] + bias[((((int)threadIdx.x) >> 5) + 16)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 38535200)] = max((conv_local[13] + bias[((((int)threadIdx.x) >> 5) + 24)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 51380256)] = max((conv_local[17] + bias[((((int)threadIdx.x) >> 5) + 32)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 64225312)] = max((conv_local[21] + bias[((((int)threadIdx.x) >> 5) + 40)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 77070368)] = max((conv_local[25] + bias[((((int)threadIdx.x) >> 5) + 48)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 89915424)] = max((conv_local[29] + bias[((((int)threadIdx.x) >> 5) + 56)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 64)] = max((conv_local[2] + bias[(((int)threadIdx.x) >> 5)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 12845120)] = max((conv_local[6] + bias[((((int)threadIdx.x) >> 5) + 8)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 25690176)] = max((conv_local[10] + bias[((((int)threadIdx.x) >> 5) + 16)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 38535232)] = max((conv_local[14] + bias[((((int)threadIdx.x) >> 5) + 24)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 51380288)] = max((conv_local[18] + bias[((((int)threadIdx.x) >> 5) + 32)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 64225344)] = max((conv_local[22] + bias[((((int)threadIdx.x) >> 5) + 40)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 77070400)] = max((conv_local[26] + bias[((((int)threadIdx.x) >> 5) + 48)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 89915456)] = max((conv_local[30] + bias[((((int)threadIdx.x) >> 5) + 56)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 96)] = max((conv_local[3] + bias[(((int)threadIdx.x) >> 5)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 12845152)] = max((conv_local[7] + bias[((((int)threadIdx.x) >> 5) + 8)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 25690208)] = max((conv_local[11] + bias[((((int)threadIdx.x) >> 5) + 16)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 38535264)] = max((conv_local[15] + bias[((((int)threadIdx.x) >> 5) + 24)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 51380320)] = max((conv_local[19] + bias[((((int)threadIdx.x) >> 5) + 32)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 64225376)] = max((conv_local[23] + bias[((((int)threadIdx.x) >> 5) + 40)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 77070432)] = max((conv_local[27] + bias[((((int)threadIdx.x) >> 5) + 48)]), 0.000000e+00f);
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 1605632) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) & 31)) + 89915488)] = max((conv_local[31] + bias[((((int)threadIdx.x) >> 5) + 56)]), 0.000000e+00f);
}


int main(int argc, char *argv[])
{
    std::string path;
    int input_size0 = 20313600;
    int input_size1 = 9408;
    int input_size2 = 64;
    int output_size0 = 102760448;

    checkCudaErrors(hipInit(0));
    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));
    hipCtx_t context;
    checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

    float *input0h, *input1h, *input2h, *output0h;
    float *input0d, *input1d, *input2d, *output0d;
    input0h = (float*)malloc(81254400);
    input1h = (float*)malloc(37632);
    input2h = (float*)malloc(256);

    hipMalloc((void **)&input0d, 81254400);
    hipMalloc((void **)&input1d, 37632);
    hipMalloc((void **)&input2d, 256);
    hipMalloc((void **)&output0d, 411041792);

    srand(1);
    for (int i = 0; i < input_size0; ++ i)
        input0h[i] = 1;
    for (int i = 0; i < input_size1; ++ i)
        input1h[i] = 1;
    for (int i = 0; i < input_size2; ++ i)
        input2h[i] = 1;

    hipMemcpy(input0d, input0h, 81254400, hipMemcpyHostToDevice);
    hipMemcpy(input1d, input1h, 37632, hipMemcpyHostToDevice);
    hipMemcpy(input2d, input2h, 256, hipMemcpyHostToDevice);

    dim3 grid(12544, 1, 1);
    dim3 block(256, 1, 1);

    for (int i = 0; i < 10; ++i)
    {
        default_function_kernel0<<<grid, block>>>((float*)input0d, (float*)input1d, (float*)output0d, (float*)input2d);
        hipDeviceSynchronize();
    }
}
